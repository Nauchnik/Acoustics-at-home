#include "hip/hip_runtime.h"
#include "gpu32.h"
#include "hip/hip_runtime_api.h"

#define BLOCKSIZE 8

#define MAX_MAT_SIZE 2048
#define MAX_FREQS 1000
#define MAX_INTERFACES 10
#define MAX_WNUMS 100

#ifndef ORD_RICH
  #define ORD_RICH 1
#endif

#ifndef FTYPE	
  #define ftype float
#else
  #define ftype FTYPE
#endif

#include "residual/bisect_gpu.h"

//#define m_CopyToGPU(d, s,  bytes){ if(hipSuccess != hipMalloc((void**) &d, bytes)) boinc_temporary_exit(60); checkCudaErrors(hipMemcpy((void*) d, (void*)s, bytes, hipMemcpyHostToDevice)); }

#define m_CopyToGPU2(s, elements, type)\
	type* g_##s;\
	checkCudaErrors(hipMalloc((void**) &g_##s, elements*sizeof(type)));\
	checkCudaErrors(hipMemcpy((void*) g_##s, (void*)s, elements*sizeof(type), hipMemcpyHostToDevice));

#define m_FreeHostAndGPU(s)\
	free(s);\
	checkCudaErrors(hipFree(g_##s));

#define m_FreeGPU(s)\
	checkCudaErrors(hipFree(g_##s));

__device__ void FillLocalArrays (
		const int tid,
		const ftype cb,
		const ftype rhob, 
		const int batch_sz, 
		const int cws_sz, 
		const ftype* cws, 
		ftype rhos[],
		ftype c1s[],
		ftype c2s[])
{
	// FIXME: magic constant!
	for (int i = 0; i < cws_sz +1; ++i)
		c1s[i] = 1500;

	for (unsigned i = 0; i < cws_sz - 1; i++)
	{
		c1s[i] = cws[i*batch_sz + tid];
		c2s[i] = cws[(i + 1)*batch_sz + tid];
		rhos[i] = 1;
	}

	c1s[cws_sz - 1] = cws[(cws_sz - 1)*batch_sz + tid];
	c2s[cws_sz - 1] = cws[(cws_sz - 1)*batch_sz + tid];
	rhos[cws_sz - 1] = 1;

	c1s[cws_sz] = cb;
	c2s[cws_sz] = cb;
	rhos[cws_sz] = rhob;
}

__device__ void FillDiagonals(
		const ftype omega,
		const ftype c[], 
		const int c_sz,
		const ftype rho[],
		const int interface_idcs[], 
		const int interface_idcs_sz,
		const ftype meshsizes[],
		ftype md[], 
		ftype ud[] /*sd*/, 
		int& mat_size)
{
	int N_points = c_sz;
	int layer_number = 0;

	ftype ld[MAX_MAT_SIZE];
	ftype dz = meshsizes[layer_number];
	for (int i = 0; i < N_points - 2; i++)
	{
		if ((layer_number < interface_idcs_sz) && (i == (interface_idcs[layer_number]-1)))
		{
			// special case of the point at the interface
			++layer_number;
			ftype dz_next = meshsizes[layer_number];
			ftype cp = c[i + 1];
			ftype dp = rho[i + 1];
			ftype cm = c[i];
			ftype dm = rho[i];
			ftype q = 1 / (dz_next * dm + dz * dp);

			ld[i] = 2 * q * dp / dz;
			// Magic!
			md[i] = -2 * q * (dz_next * dp + dz * dm) / (dz * dz_next) +
						omega * omega * q * (dz * dp * cp * cp + dz_next * dm * cm * cm) /
							(cp * cp * cm * cm);
			ud[i] = 2 * q * dm / dz_next;
			dz = dz_next;
		} 
		else 
		{
			// ordinary point
			ud[i] = (1 / (dz * dz)); 
			ld[i] = ud[i];
			md[i] = (-2 / (dz * dz) + omega * omega / (c[i + 1] * c[i + 1]));
		}
	}

	// TODO: merge me with ud-ld-md cycle and remove ld array!
	// Symmetrize the matrix
	for (int i = 0; i < N_points - 3; i++)
		ud[i] = sqrt(ud[i] * ld[i + 1]);
	mat_size = N_points - 2;
	// DIAGONALS!!!
}

__device__ Interval ComputeWavenumsLimits(
		const ftype omega, 
		const ftype c[], 
		const int c_sz)
{
	ftype cmin = c[0], cmax = c[0];
	for (int i = 0; i < c_sz; i++)
	{
		if (c[i] < cmin)
			cmin = c[i];
		if (c[i] > cmax)
			cmax = c[i];
	}
	ftype kappamax = omega / cmin;
	ftype kappamin = omega / cmax;
	return Interval {kappamin*kappamin, kappamax*kappamax};
}

__device__ void FillLayers(const int rr, 
		const int n_layers,
		const ftype* depths, 
		const ftype* rhos, 
		const ftype* c1s,
		const ftype* c2s, 
		const int* Ns_points, 
		ftype mesh[], 
		int interface_idcs[], int& interface_idcs_sz,
		ftype c[], int& c_sz, ftype rho[])
{
	c[0] = 0;
	rho[0] = 0;

	// TODO: Rewrite me, i am UGLY ((
	int n = 1; //total number of points
	ftype zp = 0;
	for (unsigned i = 0; i < n_layers; ++i)
	{
		int n_points_layer = Ns_points[i] * rr;
		ftype zc = depths[i];
		mesh[i] = (zc - zp) / n_points_layer; // dz

		c[n - 1] = c1s[i];
		rho[n - 1] = rhos[i];

		for (unsigned k = 1; k <= n_points_layer; ++k)
		{
			rho[n] = rhos[i];
			c[n] = (c1s[i] + (c2s[i] - c1s[i]) * k / n_points_layer);
			++n;
		}
		if (i < n_layers - 1)
			interface_idcs[i] = n - 1;
		zp = zc;
	}

	interface_idcs_sz = n_layers - 1;
	c_sz = n;
}

__device__ void ComputeWavenums(
		const ftype omega,
		const int n_layers,
		const int* Ns_points,
		const ftype* depths,
		const ftype rhos[],
		const ftype c1s[],
		const ftype c2s[],
		ftype wnums[],
		int& wnums_sz)
{
	// Strange things happen here...
	int  Ns_points_aligned [MAX_MAT_SIZE];
	for (int i = 0; i < n_layers; ++i)
		Ns_points_aligned[i] = 12 * (Ns_points[i] / 12);

	ftype coeff_extrap[4][4] = {
			{1,0,0,0},
			{-1, 2, 0, 0},
			{0.5, -4, 4.5, 0},
			{-1 / ftype(6), 4, -13.5, 32 / ftype(3)}};

	for (int rr = 1; rr <= ORD_RICH; ++rr)
	{
		ftype mesh [MAX_MAT_SIZE];
		int interface_idcs [MAX_INTERFACES]; 
		int interface_idcs_sz;
		ftype c [MAX_MAT_SIZE];
		int   c_sz;
		ftype rho [MAX_MAT_SIZE];
		FillLayers(rr, n_layers, depths, rhos, c1s, c2s, Ns_points_aligned, 
				mesh, interface_idcs, interface_idcs_sz, c, c_sz, rho);

		int mat_size;
		ftype md [MAX_MAT_SIZE];
		ftype sd [MAX_MAT_SIZE];
		FillDiagonals(omega, c, c_sz, rho, interface_idcs, interface_idcs_sz, mesh, 
				md, sd, mat_size);

		ftype wnums_rr [MAX_WNUMS];
		int wnums_rr_sz;
		Interval lim = ComputeWavenumsLimits(omega, c, c_sz);
		wnums_rr_sz = bisectGPU(md, sd, mat_size, lim.ll, lim.rl, wnums_rr);
		if (rr == 1) 
			wnums_sz = wnums_rr_sz;
		for (int i = 0; i < wnums_rr_sz; ++i)
			wnums[i] += (wnums_rr[i] * coeff_extrap[ORD_RICH-1][rr-1]);
	}
}

// This procedure computes MGV for a _single_ frequency
__device__ void ComputeModalGroupVelocities (
		const ftype freq,
		const int n_layers,
		const int* Ns_points,
		const ftype* depths,
		const ftype rhos[],
		const ftype c1s[],
		const ftype c2s[],
		ftype mgv[MAX_WNUMS],
		int& mgv_sz)
{
	ftype wnums1 [MAX_WNUMS] = {0}; int wnums1_sz;
	ftype wnums2 [MAX_WNUMS] = {0}; int wnums2_sz;
	// magic number for numerical differentiation procedure
	ftype deltaf = 0.05;
	ftype omega1 = 2 * LOCAL_M_PI * freq + deltaf;
	ftype omega2 = 2 * LOCAL_M_PI * freq;
	
	ComputeWavenums(omega1, n_layers, Ns_points, depths, rhos, c1s, c2s, wnums1, wnums1_sz);
	ComputeWavenums(omega2, n_layers, Ns_points, depths, rhos, c1s, c2s, wnums2, wnums2_sz);

	// Since with increase of omega the number of wave numbers
	// can only increase,  wnum2_sz <= wnum1_sz
	for (int i = 0; i < wnums2_sz; ++i)
		mgv[i] = (omega1 - omega2) / (sqrt(wnums1[i]) - sqrt(wnums2[i]));
	mgv_sz = wnums2_sz;
}

__global__ void EvalPoint_gpukernel(
		const int cws_sz, 
		const int dmaxsz,
		const ftype* cws, 
		const int* Ns_points,
		const ftype* depths,
		const ftype R, 
		const ftype tau, 
		const ftype rhob, 
		const ftype cb, 
		const ftype* freqs, 
		const int freqs_sz,
		const ftype* exp_delays,
		const int* exp_delays_sz,
		ftype* residual,
		int* n_res_global)
{
	int n_layers = cws_sz+1;
	
	const unsigned int tid = (blockIdx.x << BLOCKSIZE) + threadIdx.x;

	if (tid >= freqs_sz)
		return;
	ftype rhos[MAX_MAT_SIZE];
	ftype c1s[MAX_MAT_SIZE];
	ftype c2s[MAX_MAT_SIZE];
	FillLocalArrays(0, cb, rhob, 1, cws_sz, cws,  
			rhos, c1s, c2s);

	int n_residuals = 0;
	ftype residuals_local = 0;
	// Compute mgvs for all frequencies
	//assert (freqs_sz < MAX_FREQS);
	ftype calc_mgv[MAX_WNUMS];
	int calc_mgv_sz;
	ComputeModalGroupVelocities(freqs[tid], n_layers, Ns_points, depths, rhos, c1s, c2s, 
		calc_mgv, calc_mgv_sz);

	int min_size = calc_mgv_sz < exp_delays_sz[tid] ? 
		calc_mgv_sz : exp_delays_sz[tid];

	for (int j = 0; j < min_size; ++j) //iterate over modal velocities
	{
		ftype exp_delay = exp_delays[tid*dmaxsz + j];
		ftype calc_delay = R / calc_mgv[j];
		if (exp_delay > 0)
		{
			residuals_local += pow(exp_delay + tau - calc_delay, 2);
			++n_residuals;
		}
	}

	atomicAdd(residual, residuals_local);
	atomicAdd(n_res_global, n_residuals);

	//residual = sqrt(residuals_local/n_residuals);
}
void BisectResCalcGPU32::EvalPoint( //model
		const int dmaxsz,
		const int* Ns_points,
		const float* depths,
		const float* freqs, 
		const int freqs_sz,
		const float* exp_delays,
		const int* exp_delays_sz,
		//point
		const float R, 
		const float tau, 
		const float rhob, 
		const float cb, 
		const float* cws, 
		const int cws_sz, 
		//output
		float* residual,
		int* n_res_global)
{

	int n_layers = cws_sz+1;

	m_CopyToGPU2(cws, cws_sz, ftype);
	m_CopyToGPU2(freqs, freqs_sz, ftype);
	m_CopyToGPU2(exp_delays, dmaxsz*freqs_sz, ftype);
	m_CopyToGPU2(exp_delays_sz, freqs_sz, int);
	m_CopyToGPU2(depths, n_layers, ftype);
	m_CopyToGPU2(Ns_points, n_layers, int);
	m_CopyToGPU2(residual, 1, ftype);
	m_CopyToGPU2(n_res_global, 1, int);

	EvalPoint_gpukernel <<< freqs_sz/(1<<BLOCKSIZE), 1<<BLOCKSIZE >>> 
		(cws_sz, 
		 dmaxsz, 
		 g_cws, 
		 g_Ns_points, 
		 g_depths, 
		 R, 
		 tau, 
		 rhob, 
		 cb, 
		 g_freqs, 
		 freqs_sz,
		 g_exp_delays,
		 g_exp_delays_sz,
		 g_residual,
		 g_n_res_global);

	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	checkCudaErrors(err);
	//printf("\n Bla");


	checkCudaErrors(hipMemcpy((void*) residual, (void*)g_residual, 
				sizeof(ftype), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy((void*) n_res_global, (void*)g_n_res_global, 
				sizeof(int), hipMemcpyDeviceToHost));
	//printf("\n Res_loc: %f %i", *residual, *n_res_global);

	m_FreeGPU(Ns_points);
	m_FreeGPU(depths);
	m_FreeGPU(exp_delays_sz);
	m_FreeGPU(exp_delays);
	m_FreeGPU(freqs);
	m_FreeGPU(cws);
	m_FreeGPU(residual);
	m_FreeGPU(n_res_global);
}

//BisectResCalcGPU32::BisectResCalcGPU32(std::string nm) { /*name_ = nm;*/ }

/*
BisectResCalcGPU32::BisectResCalcGPU32()
{ 
	
	// FIXME: Correct name assginment on constructor!!!!
	//size_t s = sizeof(ftype)*8;
	//name_ = "cpu"+std::to_string(s);
}

*/
